#include "hip/hip_runtime.h"
//===------------ omp_data.cu - NVPTX OpenMP GPU objects --------- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the data objects used on the GPU device.
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"

////////////////////////////////////////////////////////////////////////////////
// global device envrionment
////////////////////////////////////////////////////////////////////////////////

__device__ omptarget_device_environmentTy omptarget_device_environment;

////////////////////////////////////////////////////////////////////////////////
// global data holding OpenMP state information
////////////////////////////////////////////////////////////////////////////////

__device__
    omptarget_nvptx_Queue<omptarget_nvptx_ThreadPrivateContext, OMP_STATE_COUNT>
        omptarget_nvptx_device_State[MAX_SM];

__device__ omptarget_nvptx_SimpleMemoryManager
    omptarget_nvptx_simpleMemoryManager;
__device__ __shared__ uint32_t usedMemIdx;
__device__ __shared__ uint32_t usedSlotIdx;

__device__ __shared__ uint8_t parallelLevel;

// Pointer to this team's OpenMP state object
__device__ __shared__
    omptarget_nvptx_ThreadPrivateContext *omptarget_nvptx_threadPrivateContext;

////////////////////////////////////////////////////////////////////////////////
// The team master sets the outlined parallel function in this variable to
// communicate with the workers.  Since it is in shared memory, there is one
// copy of these variables for each kernel, instance, and team.
////////////////////////////////////////////////////////////////////////////////
volatile __device__ __shared__ omptarget_nvptx_WorkFn omptarget_nvptx_workFn;

////////////////////////////////////////////////////////////////////////////////
// OpenMP kernel execution parameters
////////////////////////////////////////////////////////////////////////////////
__device__ __shared__ uint32_t execution_param;

////////////////////////////////////////////////////////////////////////////////
// Data sharing state
////////////////////////////////////////////////////////////////////////////////
__device__ __shared__ DataSharingStateTy DataSharingState;

////////////////////////////////////////////////////////////////////////////////
// Scratchpad for teams reduction.
////////////////////////////////////////////////////////////////////////////////
__device__ __shared__ void *ReductionScratchpadPtr;

////////////////////////////////////////////////////////////////////////////////
// Data sharing related variables.
////////////////////////////////////////////////////////////////////////////////
__device__ __shared__ omptarget_nvptx_SharedArgs omptarget_nvptx_globalArgs;
