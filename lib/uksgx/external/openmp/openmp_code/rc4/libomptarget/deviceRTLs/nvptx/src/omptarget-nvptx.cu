#include "hip/hip_runtime.h"
//===--- omptarget-nvptx.cu - NVPTX OpenMP GPU initialization ---- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the initialization code for the GPU
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"

////////////////////////////////////////////////////////////////////////////////
// global data tables
////////////////////////////////////////////////////////////////////////////////

extern __device__
    omptarget_nvptx_Queue<omptarget_nvptx_ThreadPrivateContext, OMP_STATE_COUNT>
        omptarget_nvptx_device_State[MAX_SM];

////////////////////////////////////////////////////////////////////////////////
// init entry points
////////////////////////////////////////////////////////////////////////////////

INLINE static unsigned smid() {
  unsigned id;
  asm("mov.u32 %0, %%smid;" : "=r"(id));
  return id;
}

EXTERN void __kmpc_kernel_init_params(void *Ptr) {
  PRINT(LD_IO, "call to __kmpc_kernel_init_params with version %f\n",
        OMPTARGET_NVPTX_VERSION);

  SetTeamsReductionScratchpadPtr(Ptr);
}

EXTERN void __kmpc_kernel_init(int ThreadLimit, int16_t RequiresOMPRuntime) {
  PRINT(LD_IO, "call to __kmpc_kernel_init with version %f\n",
        OMPTARGET_NVPTX_VERSION);
  ASSERT0(LT_FUSSY, RequiresOMPRuntime,
          "Generic always requires initialized runtime.");
  setExecutionParameters(Generic, RuntimeInitialized);

  int threadIdInBlock = GetThreadIdInBlock();
  ASSERT0(LT_FUSSY, threadIdInBlock == GetMasterThreadID(),
          "__kmpc_kernel_init() must be called by team master warp only!");
  PRINT0(LD_IO, "call to __kmpc_kernel_init for master\n");

  // Get a state object from the queue.
  int slot = smid() % MAX_SM;
  usedSlotIdx = slot;
  omptarget_nvptx_threadPrivateContext =
      omptarget_nvptx_device_State[slot].Dequeue();

  // init thread private
  int threadId = GetLogicalThreadIdInBlock(/*isSPMDExecutionMode=*/false);
  omptarget_nvptx_threadPrivateContext->InitThreadPrivateContext(threadId);

  // init team context
  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  currTeamDescr.InitTeamDescr(/*isSPMDExecutionMode=*/false);
  // this thread will start execution... has to update its task ICV
  // to point to the level zero task ICV. That ICV was init in
  // InitTeamDescr()
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(
      threadId, currTeamDescr.LevelZeroTaskDescr());

  // set number of threads and thread limit in team to started value
  omptarget_nvptx_TaskDescr *currTaskDescr =
      omptarget_nvptx_threadPrivateContext->GetTopLevelTaskDescr(threadId);
  currTaskDescr->NThreads() = GetNumberOfWorkersInTeam();
  currTaskDescr->ThreadLimit() = ThreadLimit;
}

EXTERN void __kmpc_kernel_deinit(int16_t IsOMPRuntimeInitialized) {
  PRINT0(LD_IO, "call to __kmpc_kernel_deinit\n");
  ASSERT0(LT_FUSSY, IsOMPRuntimeInitialized,
          "Generic always requires initialized runtime.");
  // Enqueue omp state object for use by another team.
  int slot = usedSlotIdx;
  omptarget_nvptx_device_State[slot].Enqueue(
      omptarget_nvptx_threadPrivateContext);
  // Done with work.  Kill the workers.
  omptarget_nvptx_workFn = 0;
}

EXTERN void __kmpc_spmd_kernel_init(int ThreadLimit, int16_t RequiresOMPRuntime,
                                    int16_t RequiresDataSharing) {
  PRINT0(LD_IO, "call to __kmpc_spmd_kernel_init\n");

  if (!RequiresOMPRuntime) {
    // If OMP runtime is not required don't initialize OMP state.
    setExecutionParameters(Spmd, RuntimeUninitialized);
    if (GetThreadIdInBlock() == 0) {
      parallelLevel = 0;
      usedSlotIdx = smid() % MAX_SM;
    }
    __SYNCTHREADS();
    return;
  }
  setExecutionParameters(Spmd, RuntimeInitialized);

  //
  // Team Context Initialization.
  //
  // In SPMD mode there is no master thread so use any cuda thread for team
  // context initialization.
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    // Get a state object from the queue.
    int slot = smid() % MAX_SM;
    usedSlotIdx = slot;
    omptarget_nvptx_threadPrivateContext =
        omptarget_nvptx_device_State[slot].Dequeue();

    omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
    omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();
    // init team context
    currTeamDescr.InitTeamDescr(/*isSPMDExecutionMode=*/true);
  }
  // FIXME: use __syncthreads instead when the function copy is fixed in LLVM.
  __SYNCTHREADS();

  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();

  //
  // Initialize task descr for each thread.
  //
  omptarget_nvptx_TaskDescr *newTaskDescr =
      omptarget_nvptx_threadPrivateContext->Level1TaskDescr(threadId);
  ASSERT0(LT_FUSSY, newTaskDescr, "expected a task descr");
  newTaskDescr->InitLevelOneTaskDescr(ThreadLimit,
                                      currTeamDescr.LevelZeroTaskDescr());
  newTaskDescr->ThreadLimit() = ThreadLimit;
  // install new top descriptor
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(threadId,
                                                             newTaskDescr);

  // init thread private from init value
  PRINT(LD_PAR,
        "thread will execute parallel region with id %d in a team of "
        "%d threads\n",
        (int)newTaskDescr->ThreadId(), (int)newTaskDescr->ThreadsInTeam());

  if (RequiresDataSharing && threadId % WARPSIZE == 0) {
    // Warp master innitializes data sharing environment.
    unsigned WID = threadId / WARPSIZE;
    __kmpc_data_sharing_slot *RootS = currTeamDescr.RootS(
        WID, WID == WARPSIZE - 1);
    DataSharingState.SlotPtr[WID] = RootS;
    DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];
  }
}

EXTERN __attribute__((deprecated)) void __kmpc_spmd_kernel_deinit() {
  __kmpc_spmd_kernel_deinit_v2(isRuntimeInitialized());
}

EXTERN void __kmpc_spmd_kernel_deinit_v2(int16_t RequiresOMPRuntime) {
  // We're not going to pop the task descr stack of each thread since
  // there are no more parallel regions in SPMD mode.
  if (!RequiresOMPRuntime)
    return;

  // FIXME: use __syncthreads instead when the function copy is fixed in LLVM.
  __SYNCTHREADS();
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    // Enqueue omp state object for use by another team.
    int slot = usedSlotIdx;
    omptarget_nvptx_device_State[slot].Enqueue(
        omptarget_nvptx_threadPrivateContext);
  }
}

// Return true if the current target region is executed in SPMD mode.
EXTERN int8_t __kmpc_is_spmd_exec_mode() {
  PRINT0(LD_IO | LD_PAR, "call to __kmpc_is_spmd_exec_mode\n");
  return isSPMDMode();
}
